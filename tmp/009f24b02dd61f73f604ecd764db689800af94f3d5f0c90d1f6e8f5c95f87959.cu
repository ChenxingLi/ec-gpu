#include "hip/hip_runtime.h"
// Defines to make the code work with both, CUDA and OpenCL
#ifdef __NVCC__
  #define DEVICE __device__
  #define GLOBAL
  #define KERNEL extern "C" __global__
  #define LOCAL
  #define CONSTANT __constant__

  #define GET_GLOBAL_ID() blockIdx.x * blockDim.x + threadIdx.x
  #define GET_GROUP_ID() blockIdx.x
  #define GET_LOCAL_ID() threadIdx.x
  #define GET_LOCAL_SIZE() blockDim.x
  #define BARRIER_LOCAL() __syncthreads()

  typedef unsigned char uchar;

  #define CUDA
#else // OpenCL
  #define DEVICE
  #define GLOBAL __global
  #define KERNEL __kernel
  #define LOCAL __local
  #define CONSTANT __constant

  #define GET_GLOBAL_ID() get_global_id(0)
  #define GET_GROUP_ID() get_group_id(0)
  #define GET_LOCAL_ID() get_local_id(0)
  #define GET_LOCAL_SIZE() get_local_size(0)
  #define BARRIER_LOCAL() barrier(CLK_LOCAL_MEM_FENCE)
#endif

#ifdef __NV_CL_C_VERSION
#define OPENCL_NVIDIA
#endif

#if defined(__WinterPark__) || defined(__BeaverCreek__) || defined(__Turks__) || \
    defined(__Caicos__) || defined(__Tahiti__) || defined(__Pitcairn__) || \
    defined(__Capeverde__) || defined(__Cayman__) || defined(__Barts__) || \
    defined(__Cypress__) || defined(__Juniper__) || defined(__Redwood__) || \
    defined(__Cedar__) || defined(__ATI_RV770__) || defined(__ATI_RV730__) || \
    defined(__ATI_RV710__) || defined(__Loveland__) || defined(__GPU__) || \
    defined(__Hawaii__)
#define AMD
#endif

// Returns a * b + c + d, puts the carry in d
DEVICE ulong mac_with_carry_64(ulong a, ulong b, ulong c, ulong *d) {
  #if defined(OPENCL_NVIDIA) || defined(CUDA)
    ulong lo, hi;
    asm("mad.lo.cc.u64 %0, %2, %3, %4;\r\n"
        "madc.hi.u64 %1, %2, %3, 0;\r\n"
        "add.cc.u64 %0, %0, %5;\r\n"
        "addc.u64 %1, %1, 0;\r\n"
        : "=l"(lo), "=l"(hi) : "l"(a), "l"(b), "l"(c), "l"(*d));
    *d = hi;
    return lo;
  #else
    ulong lo = a * b + c;
    ulong hi = mad_hi(a, b, (ulong)(lo < c));
    a = lo;
    lo += *d;
    hi += (lo < a);
    *d = hi;
    return lo;
  #endif
}

// Returns a + b, puts the carry in d
DEVICE ulong add_with_carry_64(ulong a, ulong *b) {
  #if defined(OPENCL_NVIDIA) || defined(CUDA)
    ulong lo, hi;
    asm("add.cc.u64 %0, %2, %3;\r\n"
        "addc.u64 %1, 0, 0;\r\n"
        : "=l"(lo), "=l"(hi) : "l"(a), "l"(*b));
    *b = hi;
    return lo;
  #else
    ulong lo = a + *b;
    *b = lo < a;
    return lo;
  #endif
}

// Returns a * b + c + d, puts the carry in d
DEVICE uint mac_with_carry_32(uint a, uint b, uint c, uint *d) {
  ulong res = (ulong)a * b + c + *d;
  *d = res >> 32;
  return res;
}

// Returns a + b, puts the carry in b
DEVICE uint add_with_carry_32(uint a, uint *b) {
  #if defined(OPENCL_NVIDIA) || defined(CUDA)
    uint lo, hi;
    asm("add.cc.u32 %0, %2, %3;\r\n"
        "addc.u32 %1, 0, 0;\r\n"
        : "=r"(lo), "=r"(hi) : "r"(a), "r"(*b));
    *b = hi;
    return lo;
  #else
    uint lo = a + *b;
    *b = lo < a;
    return lo;
  #endif
}

// Reverse the given bits. It's used by the FFT kernel.
DEVICE uint bitreverse(uint n, uint bits) {
  uint r = 0;
  for(int i = 0; i < bits; i++) {
    r = (r << 1) | (n & 1);
    n >>= 1;
  }
  return r;
}

#ifdef CUDA
// CUDA doesn't support local buffers ("dynamic shared memory" in CUDA lingo) as function
// arguments, but only a single globally defined extern value. Use `uchar` so that it is always
// allocated by the number of bytes.
extern __shared__ uchar cuda_shared[];

typedef uint uint32_t;
typedef int  int32_t;
typedef uint limb;

DEVICE inline uint32_t add_cc(uint32_t a, uint32_t b) {
  uint32_t r;

  asm volatile ("add.cc.u32 %0, %1, %2;" : "=r"(r) : "r"(a), "r"(b));
  return r;
}

DEVICE inline uint32_t addc_cc(uint32_t a, uint32_t b) {
  uint32_t r;

  asm volatile ("addc.cc.u32 %0, %1, %2;" : "=r"(r) : "r"(a), "r"(b));
  return r;
}

DEVICE inline uint32_t addc(uint32_t a, uint32_t b) {
  uint32_t r;

  asm volatile ("addc.u32 %0, %1, %2;" : "=r"(r) : "r"(a), "r"(b));
  return r;
}


DEVICE inline uint32_t madlo(uint32_t a, uint32_t b, uint32_t c) {
  uint32_t r;

  asm volatile ("mad.lo.u32 %0, %1, %2, %3;" : "=r"(r) : "r"(a), "r"(b), "r"(c));
  return r;
}

DEVICE inline uint32_t madlo_cc(uint32_t a, uint32_t b, uint32_t c) {
  uint32_t r;

  asm volatile ("mad.lo.cc.u32 %0, %1, %2, %3;" : "=r"(r) : "r"(a), "r"(b), "r"(c));
  return r;
}

DEVICE inline uint32_t madloc_cc(uint32_t a, uint32_t b, uint32_t c) {
  uint32_t r;

  asm volatile ("madc.lo.cc.u32 %0, %1, %2, %3;" : "=r"(r) : "r"(a), "r"(b), "r"(c));
  return r;
}

DEVICE inline uint32_t madloc(uint32_t a, uint32_t b, uint32_t c) {
  uint32_t r;

  asm volatile ("madc.lo.u32 %0, %1, %2, %3;" : "=r"(r) : "r"(a), "r"(b), "r"(c));
  return r;
}

DEVICE inline uint32_t madhi(uint32_t a, uint32_t b, uint32_t c) {
  uint32_t r;

  asm volatile ("mad.hi.u32 %0, %1, %2, %3;" : "=r"(r) : "r"(a), "r"(b), "r"(c));
  return r;
}

DEVICE inline uint32_t madhi_cc(uint32_t a, uint32_t b, uint32_t c) {
  uint32_t r;

  asm volatile ("mad.hi.cc.u32 %0, %1, %2, %3;" : "=r"(r) : "r"(a), "r"(b), "r"(c));
  return r;
}

DEVICE inline uint32_t madhic_cc(uint32_t a, uint32_t b, uint32_t c) {
  uint32_t r;

  asm volatile ("madc.hi.cc.u32 %0, %1, %2, %3;" : "=r"(r) : "r"(a), "r"(b), "r"(c));
  return r;
}

DEVICE inline uint32_t madhic(uint32_t a, uint32_t b, uint32_t c) {
  uint32_t r;

  asm volatile ("madc.hi.u32 %0, %1, %2, %3;" : "=r"(r) : "r"(a), "r"(b), "r"(c));
  return r;
}

typedef struct {
  int32_t _position;
} chain_t;

DEVICE inline
void chain_init(chain_t *c) {
  c->_position = 0;
}

DEVICE inline
uint32_t chain_add(chain_t *ch, uint32_t a, uint32_t b) {
  uint32_t r;

  ch->_position++;
  if(ch->_position==1)
    r=add_cc(a, b);
  else
    r=addc_cc(a, b);
  return r;
}

DEVICE inline
uint32_t chain_madlo(chain_t *ch, uint32_t a, uint32_t b, uint32_t c) {
  uint32_t r;

  ch->_position++;
  if(ch->_position==1)
    r=madlo_cc(a, b, c);
  else
    r=madloc_cc(a, b, c);
  return r;
}

DEVICE inline
uint32_t chain_madhi(chain_t *ch, uint32_t a, uint32_t b, uint32_t c) {
  uint32_t r;

  ch->_position++;
  if(ch->_position==1)
    r=madhi_cc(a, b, c);
  else
    r=madhic_cc(a, b, c);
  return r;
}
#endif
#define ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__limb uint
#define ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS 8
#define ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMB_BITS 32
#define ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__INV 4026531839
typedef struct { ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__limb val[ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS]; } ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_;
typedef struct { ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__limb val[ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS]; } ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__repr;
CONSTANT ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__ONE = { { 1342177275, 2895524892, 2673921321, 922515093, 2021213742, 1718526831, 2584207151, 235567041 } };
CONSTANT ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__P = { { 4026531841, 1138881939, 2042196113, 674490440, 2172737629, 3092268470, 3778125865, 811880050 } };
CONSTANT ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__R2 = { { 2921426343, 465102405, 3814480355, 1409170097, 1404797061, 2353627965, 2135835813, 35049649 } };
CONSTANT ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__ZERO = { { 0, 0, 0, 0, 0, 0, 0, 0 } };
#if defined(OPENCL_NVIDIA) || defined(CUDA)

DEVICE ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__sub_nvidia(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ a, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ b) {
asm("sub.cc.u32 %0, %0, %8;\r\n"
"subc.cc.u32 %1, %1, %9;\r\n"
"subc.cc.u32 %2, %2, %10;\r\n"
"subc.cc.u32 %3, %3, %11;\r\n"
"subc.cc.u32 %4, %4, %12;\r\n"
"subc.cc.u32 %5, %5, %13;\r\n"
"subc.cc.u32 %6, %6, %14;\r\n"
"subc.u32 %7, %7, %15;\r\n"
:"+r"(a.val[0]), "+r"(a.val[1]), "+r"(a.val[2]), "+r"(a.val[3]), "+r"(a.val[4]), "+r"(a.val[5]), "+r"(a.val[6]), "+r"(a.val[7])
:"r"(b.val[0]), "r"(b.val[1]), "r"(b.val[2]), "r"(b.val[3]), "r"(b.val[4]), "r"(b.val[5]), "r"(b.val[6]), "r"(b.val[7]));
return a;
}
DEVICE ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__add_nvidia(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ a, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ b) {
asm("add.cc.u32 %0, %0, %8;\r\n"
"addc.cc.u32 %1, %1, %9;\r\n"
"addc.cc.u32 %2, %2, %10;\r\n"
"addc.cc.u32 %3, %3, %11;\r\n"
"addc.cc.u32 %4, %4, %12;\r\n"
"addc.cc.u32 %5, %5, %13;\r\n"
"addc.cc.u32 %6, %6, %14;\r\n"
"addc.u32 %7, %7, %15;\r\n"
:"+r"(a.val[0]), "+r"(a.val[1]), "+r"(a.val[2]), "+r"(a.val[3]), "+r"(a.val[4]), "+r"(a.val[5]), "+r"(a.val[6]), "+r"(a.val[7])
:"r"(b.val[0]), "r"(b.val[1]), "r"(b.val[2]), "r"(b.val[3]), "r"(b.val[4]), "r"(b.val[5]), "r"(b.val[6]), "r"(b.val[7]));
return a;
}
#endif

// FinalityLabs - 2019
// Arbitrary size prime-field arithmetic library (add, sub, mul, pow)

#define ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__BITS (ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS * ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMB_BITS)
#if ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMB_BITS == 32
  #define ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__mac_with_carry mac_with_carry_32
  #define ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__add_with_carry add_with_carry_32
#elif ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMB_BITS == 64
  #define ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__mac_with_carry mac_with_carry_64
  #define ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__add_with_carry add_with_carry_64
#endif

// Greater than or equal
DEVICE bool ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__gte(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ a, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ b) {
  for(char i = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS - 1; i >= 0; i--){
    if(a.val[i] > b.val[i])
      return true;
    if(a.val[i] < b.val[i])
      return false;
  }
  return true;
}

// Equals
DEVICE bool ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__eq(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ a, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ b) {
  for(uchar i = 0; i < ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS; i++)
    if(a.val[i] != b.val[i])
      return false;
  return true;
}

// Normal addition
#if defined(OPENCL_NVIDIA) || defined(CUDA)
  #define ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__add_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__add_nvidia
  #define ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__sub_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__sub_nvidia
#else
  DEVICE ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__add_(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ a, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ b) {
    bool carry = 0;
    for(uchar i = 0; i < ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS; i++) {
      ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__limb old = a.val[i];
      a.val[i] += b.val[i] + carry;
      carry = carry ? old >= a.val[i] : old > a.val[i];
    }
    return a;
  }
  ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__sub_(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ a, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ b) {
    bool borrow = 0;
    for(uchar i = 0; i < ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS; i++) {
      ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__limb old = a.val[i];
      a.val[i] -= b.val[i] + borrow;
      borrow = borrow ? old <= a.val[i] : old < a.val[i];
    }
    return a;
  }
#endif

// Modular subtraction
DEVICE ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__sub(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ a, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ b) {
  ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ res = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__sub_(a, b);
  if(!ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__gte(a, b)) res = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__add_(res, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__P);
  return res;
}

// Modular addition
DEVICE ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__add(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ a, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ b) {
  ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ res = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__add_(a, b);
  if(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__gte(res, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__P)) res = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__sub_(res, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__P);
  return res;
}


#ifdef CUDA
// Code based on the work from Supranational, with special thanks to Niall Emmart:
//
// We would like to acknowledge Niall Emmart at Nvidia for his significant
// contribution of concepts and code for generating efficient SASS on
// Nvidia GPUs. The following papers may be of interest:
//     Optimizing Modular Multiplication for NVIDIA's Maxwell GPUs
//     https://ieeexplore.ieee.org/document/7563271
//
//     Faster modular exponentiation using double precision floating point
//     arithmetic on the GPU
//     https://ieeexplore.ieee.org/document/8464792

DEVICE void ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__reduce(uint32_t accLow[ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS], uint32_t np0, uint32_t fq[ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS]) {
  // accLow is an IN and OUT vector
  // count must be even
  const uint32_t count = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS;
  uint32_t accHigh[ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS];
  uint32_t bucket=0, lowCarry=0, highCarry=0, q;
  int32_t  i, j;

  #pragma unroll
  for(i=0;i<count;i++)
    accHigh[i]=0;

  // bucket is used so we don't have to push a carry all the way down the line

  #pragma unroll
  for(j=0;j<count;j++) {       // main iteration
    if(j%2==0) {
      add_cc(bucket, 0xFFFFFFFF);
      accLow[0]=addc_cc(accLow[0], accHigh[1]);
      bucket=addc(0, 0);

      q=accLow[0]*np0;

      chain_t chain1;
      chain_init(&chain1);

      #pragma unroll
      for(i=0;i<count;i+=2) {
        accLow[i]=chain_madlo(&chain1, q, fq[i], accLow[i]);
        accLow[i+1]=chain_madhi(&chain1, q, fq[i], accLow[i+1]);
      }
      lowCarry=chain_add(&chain1, 0, 0);

      chain_t chain2;
      chain_init(&chain2);
      for(i=0;i<count-2;i+=2) {
        accHigh[i]=chain_madlo(&chain2, q, fq[i+1], accHigh[i+2]);    // note the shift down
        accHigh[i+1]=chain_madhi(&chain2, q, fq[i+1], accHigh[i+3]);
      }
      accHigh[i]=chain_madlo(&chain2, q, fq[i+1], highCarry);
      accHigh[i+1]=chain_madhi(&chain2, q, fq[i+1], 0);
    }
    else {
      add_cc(bucket, 0xFFFFFFFF);
      accHigh[0]=addc_cc(accHigh[0], accLow[1]);
      bucket=addc(0, 0);

      q=accHigh[0]*np0;

      chain_t chain3;
      chain_init(&chain3);
      #pragma unroll
      for(i=0;i<count;i+=2) {
        accHigh[i]=chain_madlo(&chain3, q, fq[i], accHigh[i]);
        accHigh[i+1]=chain_madhi(&chain3, q, fq[i], accHigh[i+1]);
      }
      highCarry=chain_add(&chain3, 0, 0);

      chain_t chain4;
      chain_init(&chain4);
      for(i=0;i<count-2;i+=2) {
        accLow[i]=chain_madlo(&chain4, q, fq[i+1], accLow[i+2]);    // note the shift down
        accLow[i+1]=chain_madhi(&chain4, q, fq[i+1], accLow[i+3]);
      }
      accLow[i]=chain_madlo(&chain4, q, fq[i+1], lowCarry);
      accLow[i+1]=chain_madhi(&chain4, q, fq[i+1], 0);
    }
  }

  // at this point, accHigh needs to be shifted back a word and added to accLow
  // we'll use one other trick.  Bucket is either 0 or 1 at this point, so we
  // can just push it into the carry chain.

  chain_t chain5;
  chain_init(&chain5);
  chain_add(&chain5, bucket, 0xFFFFFFFF);    // push the carry into the chain
  #pragma unroll
  for(i=0;i<count-1;i++)
    accLow[i]=chain_add(&chain5, accLow[i], accHigh[i+1]);
  accLow[i]=chain_add(&chain5, accLow[i], highCarry);
}

// Requirement: yLimbs >= xLimbs
DEVICE inline
void ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__mult_v1(uint32_t *x, uint32_t *y, uint32_t *xy) {
  const uint32_t xLimbs  = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS;
  const uint32_t yLimbs  = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS;
  const uint32_t xyLimbs = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS * 2;
  uint32_t temp[ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS * 2];
  uint32_t carry = 0;

  #pragma unroll
  for (int32_t i = 0; i < xyLimbs; i++) {
    temp[i] = 0;
  }

  #pragma unroll
  for (int32_t i = 0; i < xLimbs; i++) {
    chain_t chain1;
    chain_init(&chain1);
    #pragma unroll
    for (int32_t j = 0; j < yLimbs; j++) {
      if ((i + j) % 2 == 1) {
        temp[i + j - 1] = chain_madlo(&chain1, x[i], y[j], temp[i + j - 1]);
        temp[i + j]     = chain_madhi(&chain1, x[i], y[j], temp[i + j]);
      }
    }
    if (i % 2 == 1) {
      temp[i + yLimbs - 1] = chain_add(&chain1, 0, 0);
    }
  }

  #pragma unroll
  for (int32_t i = xyLimbs - 1; i > 0; i--) {
    temp[i] = temp[i - 1];
  }
  temp[0] = 0;

  #pragma unroll
  for (int32_t i = 0; i < xLimbs; i++) {
    chain_t chain2;
    chain_init(&chain2);

    #pragma unroll
    for (int32_t j = 0; j < yLimbs; j++) {
      if ((i + j) % 2 == 0) {
        temp[i + j]     = chain_madlo(&chain2, x[i], y[j], temp[i + j]);
        temp[i + j + 1] = chain_madhi(&chain2, x[i], y[j], temp[i + j + 1]);
      }
    }
    if ((i + yLimbs) % 2 == 0 && i != yLimbs - 1) {
      temp[i + yLimbs]     = chain_add(&chain2, temp[i + yLimbs], carry);
      temp[i + yLimbs + 1] = chain_add(&chain2, temp[i + yLimbs + 1], 0);
      carry = chain_add(&chain2, 0, 0);
    }
    if ((i + yLimbs) % 2 == 1 && i != yLimbs - 1) {
      carry = chain_add(&chain2, carry, 0);
    }
  }

  #pragma unroll
  for(int32_t i = 0; i < xyLimbs; i++) {
    xy[i] = temp[i];
  }
}

DEVICE ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__mul_nvidia(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ a, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ b) {
  // Perform full multiply
  limb ab[2 * ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS];
  ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__mult_v1(a.val, b.val, ab);

  uint32_t io[ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS];
  #pragma unroll
  for(int i=0;i<ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS;i++) {
    io[i]=ab[i];
  }
  ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__reduce(io, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__INV, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__P.val);

  // Add io to the upper words of ab
  ab[ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS] = add_cc(ab[ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS], io[0]);
  int j;
  #pragma unroll
  for (j = 1; j < ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS - 1; j++) {
    ab[j + ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS] = addc_cc(ab[j + ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS], io[j]);
  }
  ab[2 * ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS - 1] = addc(ab[2 * ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS - 1], io[ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS - 1]);

  ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ r;
  #pragma unroll
  for (int i = 0; i < ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS; i++) {
    r.val[i] = ab[i + ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS];
  }

  if (ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__gte(r, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__P)) {
    r = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__sub_(r, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__P);
  }

  return r;
}

#endif

// Modular multiplication
DEVICE ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__mul_default(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ a, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ b) {
  /* CIOS Montgomery multiplication, inspired from Tolga Acar's thesis:
   * https://www.microsoft.com/en-us/research/wp-content/uploads/1998/06/97Acar.pdf
   * Learn more:
   * https://en.wikipedia.org/wiki/Montgomery_modular_multiplication
   * https://alicebob.cryptoland.net/understanding-the-montgomery-reduction-algorithm/
   */
  ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__limb t[ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS + 2] = {0};
  for(uchar i = 0; i < ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS; i++) {
    ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__limb carry = 0;
    for(uchar j = 0; j < ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS; j++)
      t[j] = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__mac_with_carry(a.val[j], b.val[i], t[j], &carry);
    t[ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS] = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__add_with_carry(t[ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS], &carry);
    t[ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS + 1] = carry;

    carry = 0;
    ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__limb m = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__INV * t[0];
    ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__mac_with_carry(m, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__P.val[0], t[0], &carry);
    for(uchar j = 1; j < ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS; j++)
      t[j - 1] = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__mac_with_carry(m, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__P.val[j], t[j], &carry);

    t[ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS - 1] = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__add_with_carry(t[ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS], &carry);
    t[ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS] = t[ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS + 1] + carry;
  }

  ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ result;
  for(uchar i = 0; i < ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS; i++) result.val[i] = t[i];

  if(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__gte(result, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__P)) result = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__sub_(result, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__P);

  return result;
}

#ifdef CUDA
DEVICE ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__mul(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ a, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ b) {
  return ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__mul_nvidia(a, b);
}
#else
DEVICE ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__mul(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ a, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ b) {
  return ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__mul_default(a, b);
}
#endif

// Squaring is a special case of multiplication which can be done ~1.5x faster.
// https://stackoverflow.com/a/16388571/1348497
DEVICE ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__sqr(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ a) {
  return ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__mul(a, a);
}

// Left-shift the limbs by one bit and subtract by modulus in case of overflow.
// Faster version of ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__add(a, a)
DEVICE ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__double(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ a) {
  for(uchar i = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS - 1; i >= 1; i--)
    a.val[i] = (a.val[i] << 1) | (a.val[i - 1] >> (ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMB_BITS - 1));
  a.val[0] <<= 1;
  if(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__gte(a, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__P)) a = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__sub_(a, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__P);
  return a;
}

// Modular exponentiation (Exponentiation by Squaring)
// https://en.wikipedia.org/wiki/Exponentiation_by_squaring
DEVICE ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__pow(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ base, uint exponent) {
  ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ res = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__ONE;
  while(exponent > 0) {
    if (exponent & 1)
      res = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__mul(res, base);
    exponent = exponent >> 1;
    base = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__sqr(base);
  }
  return res;
}


// Store squares of the base in a lookup table for faster evaluation.
DEVICE ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__pow_lookup(GLOBAL ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ *bases, uint exponent) {
  ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ res = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__ONE;
  uint i = 0;
  while(exponent > 0) {
    if (exponent & 1)
      res = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__mul(res, bases[i]);
    exponent = exponent >> 1;
    i++;
  }
  return res;
}


DEVICE ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__mont(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__repr a) {
  #ifdef CUDA
    ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ input = reinterpret_cast<ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_&>(a);  
  #else
    ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ input = * (ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ *) &a;
  #endif

  return ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__mul(input, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__R2);
}

DEVICE ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__repr ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__unmont(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ a) {
  ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ one = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__ZERO;
  one.val[0] = 1;
  ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ unmont = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__mul(a, one);

  
  #ifdef CUDA
    ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__repr answer = reinterpret_cast<ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__repr&>(unmont);  
  #else
    ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__repr answer = * (ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__repr *) &unmont;
  #endif
  return answer;
}

// Get `i`th bit (From most significant digit) of the field.
DEVICE bool ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__get_bit(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__repr l, uint i) {
  return (l.val[ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMBS - 1 - i / ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMB_BITS] >> (ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMB_BITS - 1 - (i % ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__LIMB_BITS))) & 1;
}

// Get `window` consecutive bits, (Starting from `skip`th bit) from the field.
DEVICE uint ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__get_bits(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__repr l, uint skip, uint window) {
  uint ret = 0;
  for(uint i = 0; i < window; i++) {
    ret <<= 1;
    ret |= ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__get_bit(l, skip + i);
  }
  return ret;
}

#define ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__limb uint
#define ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS 8
#define ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMB_BITS 32
#define ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__INV 3834012553
typedef struct { ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__limb val[ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS]; } ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_;
typedef struct { ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__limb val[ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS]; } ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__repr;
CONSTANT ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__ONE = { { 3314486685, 3546104717, 4123462461, 175696680, 2021213740, 1718526831, 2584207151, 235567041 } };
CONSTANT ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__P = { { 3632069959, 1008765974, 1752287885, 2541841041, 2172737629, 3092268470, 3778125865, 811880050 } };
CONSTANT ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__R2 = { { 1401617033, 4079811675, 3561292283, 3051821329, 172064758, 1202396927, 3401069855, 114859889 } };
CONSTANT ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__ZERO = { { 0, 0, 0, 0, 0, 0, 0, 0 } };
#if defined(OPENCL_NVIDIA) || defined(CUDA)

DEVICE ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sub_nvidia(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ a, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ b) {
asm("sub.cc.u32 %0, %0, %8;\r\n"
"subc.cc.u32 %1, %1, %9;\r\n"
"subc.cc.u32 %2, %2, %10;\r\n"
"subc.cc.u32 %3, %3, %11;\r\n"
"subc.cc.u32 %4, %4, %12;\r\n"
"subc.cc.u32 %5, %5, %13;\r\n"
"subc.cc.u32 %6, %6, %14;\r\n"
"subc.u32 %7, %7, %15;\r\n"
:"+r"(a.val[0]), "+r"(a.val[1]), "+r"(a.val[2]), "+r"(a.val[3]), "+r"(a.val[4]), "+r"(a.val[5]), "+r"(a.val[6]), "+r"(a.val[7])
:"r"(b.val[0]), "r"(b.val[1]), "r"(b.val[2]), "r"(b.val[3]), "r"(b.val[4]), "r"(b.val[5]), "r"(b.val[6]), "r"(b.val[7]));
return a;
}
DEVICE ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__add_nvidia(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ a, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ b) {
asm("add.cc.u32 %0, %0, %8;\r\n"
"addc.cc.u32 %1, %1, %9;\r\n"
"addc.cc.u32 %2, %2, %10;\r\n"
"addc.cc.u32 %3, %3, %11;\r\n"
"addc.cc.u32 %4, %4, %12;\r\n"
"addc.cc.u32 %5, %5, %13;\r\n"
"addc.cc.u32 %6, %6, %14;\r\n"
"addc.u32 %7, %7, %15;\r\n"
:"+r"(a.val[0]), "+r"(a.val[1]), "+r"(a.val[2]), "+r"(a.val[3]), "+r"(a.val[4]), "+r"(a.val[5]), "+r"(a.val[6]), "+r"(a.val[7])
:"r"(b.val[0]), "r"(b.val[1]), "r"(b.val[2]), "r"(b.val[3]), "r"(b.val[4]), "r"(b.val[5]), "r"(b.val[6]), "r"(b.val[7]));
return a;
}
#endif

// FinalityLabs - 2019
// Arbitrary size prime-field arithmetic library (add, sub, mul, pow)

#define ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__BITS (ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS * ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMB_BITS)
#if ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMB_BITS == 32
  #define ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__mac_with_carry mac_with_carry_32
  #define ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__add_with_carry add_with_carry_32
#elif ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMB_BITS == 64
  #define ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__mac_with_carry mac_with_carry_64
  #define ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__add_with_carry add_with_carry_64
#endif

// Greater than or equal
DEVICE bool ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__gte(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ a, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ b) {
  for(char i = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS - 1; i >= 0; i--){
    if(a.val[i] > b.val[i])
      return true;
    if(a.val[i] < b.val[i])
      return false;
  }
  return true;
}

// Equals
DEVICE bool ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__eq(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ a, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ b) {
  for(uchar i = 0; i < ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS; i++)
    if(a.val[i] != b.val[i])
      return false;
  return true;
}

// Normal addition
#if defined(OPENCL_NVIDIA) || defined(CUDA)
  #define ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__add_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__add_nvidia
  #define ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sub_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sub_nvidia
#else
  DEVICE ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__add_(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ a, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ b) {
    bool carry = 0;
    for(uchar i = 0; i < ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS; i++) {
      ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__limb old = a.val[i];
      a.val[i] += b.val[i] + carry;
      carry = carry ? old >= a.val[i] : old > a.val[i];
    }
    return a;
  }
  ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sub_(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ a, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ b) {
    bool borrow = 0;
    for(uchar i = 0; i < ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS; i++) {
      ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__limb old = a.val[i];
      a.val[i] -= b.val[i] + borrow;
      borrow = borrow ? old <= a.val[i] : old < a.val[i];
    }
    return a;
  }
#endif

// Modular subtraction
DEVICE ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sub(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ a, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ b) {
  ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ res = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sub_(a, b);
  if(!ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__gte(a, b)) res = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__add_(res, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__P);
  return res;
}

// Modular addition
DEVICE ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__add(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ a, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ b) {
  ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ res = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__add_(a, b);
  if(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__gte(res, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__P)) res = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sub_(res, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__P);
  return res;
}


#ifdef CUDA
// Code based on the work from Supranational, with special thanks to Niall Emmart:
//
// We would like to acknowledge Niall Emmart at Nvidia for his significant
// contribution of concepts and code for generating efficient SASS on
// Nvidia GPUs. The following papers may be of interest:
//     Optimizing Modular Multiplication for NVIDIA's Maxwell GPUs
//     https://ieeexplore.ieee.org/document/7563271
//
//     Faster modular exponentiation using double precision floating point
//     arithmetic on the GPU
//     https://ieeexplore.ieee.org/document/8464792

DEVICE void ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__reduce(uint32_t accLow[ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS], uint32_t np0, uint32_t fq[ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS]) {
  // accLow is an IN and OUT vector
  // count must be even
  const uint32_t count = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS;
  uint32_t accHigh[ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS];
  uint32_t bucket=0, lowCarry=0, highCarry=0, q;
  int32_t  i, j;

  #pragma unroll
  for(i=0;i<count;i++)
    accHigh[i]=0;

  // bucket is used so we don't have to push a carry all the way down the line

  #pragma unroll
  for(j=0;j<count;j++) {       // main iteration
    if(j%2==0) {
      add_cc(bucket, 0xFFFFFFFF);
      accLow[0]=addc_cc(accLow[0], accHigh[1]);
      bucket=addc(0, 0);

      q=accLow[0]*np0;

      chain_t chain1;
      chain_init(&chain1);

      #pragma unroll
      for(i=0;i<count;i+=2) {
        accLow[i]=chain_madlo(&chain1, q, fq[i], accLow[i]);
        accLow[i+1]=chain_madhi(&chain1, q, fq[i], accLow[i+1]);
      }
      lowCarry=chain_add(&chain1, 0, 0);

      chain_t chain2;
      chain_init(&chain2);
      for(i=0;i<count-2;i+=2) {
        accHigh[i]=chain_madlo(&chain2, q, fq[i+1], accHigh[i+2]);    // note the shift down
        accHigh[i+1]=chain_madhi(&chain2, q, fq[i+1], accHigh[i+3]);
      }
      accHigh[i]=chain_madlo(&chain2, q, fq[i+1], highCarry);
      accHigh[i+1]=chain_madhi(&chain2, q, fq[i+1], 0);
    }
    else {
      add_cc(bucket, 0xFFFFFFFF);
      accHigh[0]=addc_cc(accHigh[0], accLow[1]);
      bucket=addc(0, 0);

      q=accHigh[0]*np0;

      chain_t chain3;
      chain_init(&chain3);
      #pragma unroll
      for(i=0;i<count;i+=2) {
        accHigh[i]=chain_madlo(&chain3, q, fq[i], accHigh[i]);
        accHigh[i+1]=chain_madhi(&chain3, q, fq[i], accHigh[i+1]);
      }
      highCarry=chain_add(&chain3, 0, 0);

      chain_t chain4;
      chain_init(&chain4);
      for(i=0;i<count-2;i+=2) {
        accLow[i]=chain_madlo(&chain4, q, fq[i+1], accLow[i+2]);    // note the shift down
        accLow[i+1]=chain_madhi(&chain4, q, fq[i+1], accLow[i+3]);
      }
      accLow[i]=chain_madlo(&chain4, q, fq[i+1], lowCarry);
      accLow[i+1]=chain_madhi(&chain4, q, fq[i+1], 0);
    }
  }

  // at this point, accHigh needs to be shifted back a word and added to accLow
  // we'll use one other trick.  Bucket is either 0 or 1 at this point, so we
  // can just push it into the carry chain.

  chain_t chain5;
  chain_init(&chain5);
  chain_add(&chain5, bucket, 0xFFFFFFFF);    // push the carry into the chain
  #pragma unroll
  for(i=0;i<count-1;i++)
    accLow[i]=chain_add(&chain5, accLow[i], accHigh[i+1]);
  accLow[i]=chain_add(&chain5, accLow[i], highCarry);
}

// Requirement: yLimbs >= xLimbs
DEVICE inline
void ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__mult_v1(uint32_t *x, uint32_t *y, uint32_t *xy) {
  const uint32_t xLimbs  = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS;
  const uint32_t yLimbs  = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS;
  const uint32_t xyLimbs = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS * 2;
  uint32_t temp[ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS * 2];
  uint32_t carry = 0;

  #pragma unroll
  for (int32_t i = 0; i < xyLimbs; i++) {
    temp[i] = 0;
  }

  #pragma unroll
  for (int32_t i = 0; i < xLimbs; i++) {
    chain_t chain1;
    chain_init(&chain1);
    #pragma unroll
    for (int32_t j = 0; j < yLimbs; j++) {
      if ((i + j) % 2 == 1) {
        temp[i + j - 1] = chain_madlo(&chain1, x[i], y[j], temp[i + j - 1]);
        temp[i + j]     = chain_madhi(&chain1, x[i], y[j], temp[i + j]);
      }
    }
    if (i % 2 == 1) {
      temp[i + yLimbs - 1] = chain_add(&chain1, 0, 0);
    }
  }

  #pragma unroll
  for (int32_t i = xyLimbs - 1; i > 0; i--) {
    temp[i] = temp[i - 1];
  }
  temp[0] = 0;

  #pragma unroll
  for (int32_t i = 0; i < xLimbs; i++) {
    chain_t chain2;
    chain_init(&chain2);

    #pragma unroll
    for (int32_t j = 0; j < yLimbs; j++) {
      if ((i + j) % 2 == 0) {
        temp[i + j]     = chain_madlo(&chain2, x[i], y[j], temp[i + j]);
        temp[i + j + 1] = chain_madhi(&chain2, x[i], y[j], temp[i + j + 1]);
      }
    }
    if ((i + yLimbs) % 2 == 0 && i != yLimbs - 1) {
      temp[i + yLimbs]     = chain_add(&chain2, temp[i + yLimbs], carry);
      temp[i + yLimbs + 1] = chain_add(&chain2, temp[i + yLimbs + 1], 0);
      carry = chain_add(&chain2, 0, 0);
    }
    if ((i + yLimbs) % 2 == 1 && i != yLimbs - 1) {
      carry = chain_add(&chain2, carry, 0);
    }
  }

  #pragma unroll
  for(int32_t i = 0; i < xyLimbs; i++) {
    xy[i] = temp[i];
  }
}

DEVICE ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__mul_nvidia(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ a, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ b) {
  // Perform full multiply
  limb ab[2 * ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS];
  ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__mult_v1(a.val, b.val, ab);

  uint32_t io[ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS];
  #pragma unroll
  for(int i=0;i<ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS;i++) {
    io[i]=ab[i];
  }
  ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__reduce(io, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__INV, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__P.val);

  // Add io to the upper words of ab
  ab[ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS] = add_cc(ab[ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS], io[0]);
  int j;
  #pragma unroll
  for (j = 1; j < ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS - 1; j++) {
    ab[j + ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS] = addc_cc(ab[j + ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS], io[j]);
  }
  ab[2 * ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS - 1] = addc(ab[2 * ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS - 1], io[ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS - 1]);

  ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ r;
  #pragma unroll
  for (int i = 0; i < ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS; i++) {
    r.val[i] = ab[i + ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS];
  }

  if (ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__gte(r, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__P)) {
    r = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sub_(r, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__P);
  }

  return r;
}

#endif

// Modular multiplication
DEVICE ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__mul_default(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ a, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ b) {
  /* CIOS Montgomery multiplication, inspired from Tolga Acar's thesis:
   * https://www.microsoft.com/en-us/research/wp-content/uploads/1998/06/97Acar.pdf
   * Learn more:
   * https://en.wikipedia.org/wiki/Montgomery_modular_multiplication
   * https://alicebob.cryptoland.net/understanding-the-montgomery-reduction-algorithm/
   */
  ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__limb t[ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS + 2] = {0};
  for(uchar i = 0; i < ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS; i++) {
    ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__limb carry = 0;
    for(uchar j = 0; j < ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS; j++)
      t[j] = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__mac_with_carry(a.val[j], b.val[i], t[j], &carry);
    t[ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS] = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__add_with_carry(t[ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS], &carry);
    t[ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS + 1] = carry;

    carry = 0;
    ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__limb m = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__INV * t[0];
    ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__mac_with_carry(m, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__P.val[0], t[0], &carry);
    for(uchar j = 1; j < ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS; j++)
      t[j - 1] = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__mac_with_carry(m, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__P.val[j], t[j], &carry);

    t[ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS - 1] = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__add_with_carry(t[ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS], &carry);
    t[ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS] = t[ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS + 1] + carry;
  }

  ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ result;
  for(uchar i = 0; i < ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS; i++) result.val[i] = t[i];

  if(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__gte(result, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__P)) result = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sub_(result, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__P);

  return result;
}

#ifdef CUDA
DEVICE ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__mul(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ a, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ b) {
  return ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__mul_nvidia(a, b);
}
#else
DEVICE ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__mul(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ a, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ b) {
  return ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__mul_default(a, b);
}
#endif

// Squaring is a special case of multiplication which can be done ~1.5x faster.
// https://stackoverflow.com/a/16388571/1348497
DEVICE ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sqr(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ a) {
  return ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__mul(a, a);
}

// Left-shift the limbs by one bit and subtract by modulus in case of overflow.
// Faster version of ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__add(a, a)
DEVICE ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__double(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ a) {
  for(uchar i = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS - 1; i >= 1; i--)
    a.val[i] = (a.val[i] << 1) | (a.val[i - 1] >> (ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMB_BITS - 1));
  a.val[0] <<= 1;
  if(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__gte(a, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__P)) a = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sub_(a, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__P);
  return a;
}

// Modular exponentiation (Exponentiation by Squaring)
// https://en.wikipedia.org/wiki/Exponentiation_by_squaring
DEVICE ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__pow(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ base, uint exponent) {
  ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ res = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__ONE;
  while(exponent > 0) {
    if (exponent & 1)
      res = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__mul(res, base);
    exponent = exponent >> 1;
    base = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sqr(base);
  }
  return res;
}


// Store squares of the base in a lookup table for faster evaluation.
DEVICE ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__pow_lookup(GLOBAL ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ *bases, uint exponent) {
  ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ res = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__ONE;
  uint i = 0;
  while(exponent > 0) {
    if (exponent & 1)
      res = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__mul(res, bases[i]);
    exponent = exponent >> 1;
    i++;
  }
  return res;
}


DEVICE ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__mont(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__repr a) {
  #ifdef CUDA
    ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ input = reinterpret_cast<ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_&>(a);  
  #else
    ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ input = * (ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ *) &a;
  #endif

  return ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__mul(input, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__R2);
}

DEVICE ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__repr ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__unmont(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ a) {
  ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ one = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__ZERO;
  one.val[0] = 1;
  ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ unmont = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__mul(a, one);

  
  #ifdef CUDA
    ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__repr answer = reinterpret_cast<ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__repr&>(unmont);  
  #else
    ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__repr answer = * (ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__repr *) &unmont;
  #endif
  return answer;
}

// Get `i`th bit (From most significant digit) of the field.
DEVICE bool ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__get_bit(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__repr l, uint i) {
  return (l.val[ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMBS - 1 - i / ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMB_BITS] >> (ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMB_BITS - 1 - (i % ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__LIMB_BITS))) & 1;
}

// Get `window` consecutive bits, (Starting from `skip`th bit) from the field.
DEVICE uint ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__get_bits(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__repr l, uint skip, uint window) {
  uint ret = 0;
  for(uint i = 0; i < window; i++) {
    ret <<= 1;
    ret |= ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__get_bit(l, skip + i);
  }
  return ret;
}





// Elliptic curve operations (Short Weierstrass Jacobian form)

#define ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__ZERO ((ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__jacobian){ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__ZERO, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__ONE, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__ZERO})

typedef struct {
  ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ x;
  ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ y;
} ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__affine;

typedef struct {
  ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ x;
  ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ y;
  ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ z;
} ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__jacobian;

// http://www.hyperelliptic.org/EFD/g1p/auto-shortw-jacobian-0.html#doubling-dbl-2009-l
DEVICE ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__jacobian ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__double(ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__jacobian inp) {
  const ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ local_zero = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__ZERO;
  if(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__eq(inp.z, local_zero)) {
      return inp;
  }

  const ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ a = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sqr(inp.x); // A = X1^2
  const ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ b = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sqr(inp.y); // B = Y1^2
  ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ c = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sqr(b); // C = B^2

  // D = 2*((X1+B)2-A-C)
  ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ d = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__add(inp.x, b);
  d = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sqr(d); d = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sub(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sub(d, a), c); d = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__double(d);

  const ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ e = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__add(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__double(a), a); // E = 3*A
  const ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ f = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sqr(e);

  inp.z = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__mul(inp.y, inp.z); inp.z = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__double(inp.z); // Z3 = 2*Y1*Z1
  inp.x = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sub(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sub(f, d), d); // X3 = F-2*D

  // Y3 = E*(D-X3)-8*C
  c = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__double(c); c = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__double(c); c = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__double(c);
  inp.y = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sub(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__mul(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sub(d, inp.x), e), c);

  return inp;
}

// http://www.hyperelliptic.org/EFD/g1p/auto-shortw-jacobian-0.html#addition-madd-2007-bl
DEVICE ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__jacobian ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__add_mixed(ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__jacobian a, ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__affine b) {
  const ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ local_zero = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__ZERO;
  if(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__eq(a.z, local_zero)) {
    const ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ local_one = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__ONE;
    a.x = b.x;
    a.y = b.y;
    a.z = local_one;
    return a;
  }

  const ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ z1z1 = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sqr(a.z);
  const ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ u2 = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__mul(b.x, z1z1);
  const ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ s2 = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__mul(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__mul(b.y, a.z), z1z1);

  if(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__eq(a.x, u2) && ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__eq(a.y, s2)) {
      return ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__double(a);
  }

  const ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ h = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sub(u2, a.x); // H = U2-X1
  const ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ hh = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sqr(h); // HH = H^2
  ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ i = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__double(hh); i = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__double(i); // I = 4*HH
  ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ j = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__mul(h, i); // J = H*I
  ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ r = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sub(s2, a.y); r = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__double(r); // r = 2*(S2-Y1)
  const ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ v = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__mul(a.x, i);

  ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__jacobian ret;

  // X3 = r^2 - J - 2*V
  ret.x = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sub(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sub(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sqr(r), j), ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__double(v));

  // Y3 = r*(V-X3)-2*Y1*J
  j = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__mul(a.y, j); j = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__double(j);
  ret.y = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sub(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__mul(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sub(v, ret.x), r), j);

  // Z3 = (Z1+H)^2-Z1Z1-HH
  ret.z = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__add(a.z, h); ret.z = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sub(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sub(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sqr(ret.z), z1z1), hh);
  return ret;
}

// http://www.hyperelliptic.org/EFD/g1p/auto-shortw-jacobian-0.html#addition-add-2007-bl
DEVICE ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__jacobian ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__add(ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__jacobian a, ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__jacobian b) {

  const ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ local_zero = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__ZERO;
  if(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__eq(a.z, local_zero)) return b;
  if(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__eq(b.z, local_zero)) return a;

  const ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ z1z1 = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sqr(a.z); // Z1Z1 = Z1^2
  const ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ z2z2 = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sqr(b.z); // Z2Z2 = Z2^2
  const ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ u1 = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__mul(a.x, z2z2); // U1 = X1*Z2Z2
  const ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ u2 = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__mul(b.x, z1z1); // U2 = X2*Z1Z1
  ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ s1 = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__mul(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__mul(a.y, b.z), z2z2); // S1 = Y1*Z2*Z2Z2
  const ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ s2 = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__mul(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__mul(b.y, a.z), z1z1); // S2 = Y2*Z1*Z1Z1

  if(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__eq(u1, u2) && ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__eq(s1, s2))
    return ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__double(a);
  else {
    const ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ h = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sub(u2, u1); // H = U2-U1
    ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ i = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__double(h); i = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sqr(i); // I = (2*H)^2
    const ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ j = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__mul(h, i); // J = H*I
    ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ r = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sub(s2, s1); r = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__double(r); // r = 2*(S2-S1)
    const ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4_ v = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__mul(u1, i); // V = U1*I
    a.x = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sub(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sub(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sub(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sqr(r), j), v), v); // X3 = r^2 - J - 2*V

    // Y3 = r*(V - X3) - 2*S1*J
    a.y = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__mul(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sub(v, a.x), r);
    s1 = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__mul(s1, j); s1 = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__double(s1); // S1 = S1 * J * 2
    a.y = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sub(a.y, s1);

    // Z3 = ((Z1+Z2)^2 - Z1Z1 - Z2Z2)*H
    a.z = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__add(a.z, b.z); a.z = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sqr(a.z);
    a.z = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sub(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sub(a.z, z1z1), z2z2);
    a.z = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__mul(a.z, h);

    return a;
  }
}

DEVICE ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__jacobian ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__neg(ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__jacobian a) {
  a.y = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__sub(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fq__FqConfig__4___4__ZERO, a.y);
  return a;
}

DEVICE ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__jacobian ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__sub(ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__jacobian a, ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__jacobian b) {
  return ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__add(a, ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__neg(b));
}

DEVICE ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__jacobian ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__mul_exponent(ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__jacobian base, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__repr exp) {
  ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__jacobian res = ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__ZERO;
  for(uint i = 0; i < ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__BITS; i++) {
    res = ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__double(res);
    bool exp_bit_i = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__get_bit(exp, i);
    if(exp_bit_i) res = ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__add(res, base);
  }
  return res;
}

DEVICE ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__jacobian ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__mul(ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__jacobian base, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4_ exp) {
  return ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__mul_exponent(base, ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__unmont(exp));
}






/*
 * Same multiexp algorithm used in Bellman, with some modifications.
 * https://github.com/zkcrypto/bellman/blob/10c5010fd9c2ca69442dc9775ea271e286e776d8/src/multiexp.rs#L174
 * The CPU version of multiexp parallelism is done by dividing the exponent
 * values into smaller windows, and then applying a sequence of rounds to each
 * window. The GPU kernel not only assigns a thread to each window but also
 * divides the bases into several groups which highly increases the number of
 * threads running in parallel for calculating a multiexp instance.
 */

KERNEL void ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__multiexp(
    GLOBAL ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__affine *bases,
    GLOBAL ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__jacobian *buckets,
    GLOBAL ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__jacobian *results,
    GLOBAL ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__repr *exps,
    uint n,
    uint num_groups,
    uint num_windows,
    uint window_size) {

  // We have `num_windows` * `num_groups` threads per multiexp.
  const uint gid = GET_GLOBAL_ID();
  if(gid >= num_windows * num_groups) return;

  // We have (2^window_size - 1) buckets.
  const uint bucket_len = ((1 << window_size) - 1);

  // Each thread has its own set of buckets in global memory.
  buckets += bucket_len * gid;

  const ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__jacobian local_zero = ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__ZERO;
  for(uint i = 0; i < bucket_len; i++) buckets[i] = local_zero;

  // Num of elements in each group. Round the number up (ceil).
  const uint len = (n + num_groups - 1) / num_groups;

  // This thread runs the multiexp algorithm on elements from `nstart` to `nend`
  // on the window [`bits`, `bits` + `w`)
  const uint nstart = len * (gid / num_windows);
  const uint nend = min(nstart + len, n);
  const uint bits = (gid % num_windows) * window_size;
  const ushort w = min((ushort)window_size, (ushort)(ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__BITS - bits));

  ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__jacobian res = ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__ZERO;
  for(uint i = nstart; i < nend; i++) {
    uint ind = ag_types__impls__ark_ff__fields__models__fp__Fp_ark_ff__fields__models__fp__montgomery_backend__MontBackend_ark_bn254__fields__fr__FrConfig__4___4__get_bits(exps[i], bits, w);

    #if defined(OPENCL_NVIDIA) || defined(CUDA)
      // O_o, weird optimization, having a single special case makes it
      // tremendously faster!
      // 511 is chosen because it's half of the maximum bucket len, but
      // any other number works... Bigger indices seems to be better...
      if(ind == 511) buckets[510] = ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__add_mixed(buckets[510], bases[i]);
      else if(ind--) buckets[ind] = ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__add_mixed(buckets[ind], bases[i]);
    #else
      if(ind--) buckets[ind] = ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__add_mixed(buckets[ind], bases[i]);
    #endif
  }

  // Summation by parts
  // e.g. 3a + 2b + 1c = a +
  //                    (a) + b +
  //                    ((a) + b) + c
  ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__jacobian acc = ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__ZERO;
  for(int j = bucket_len - 1; j >= 0; j--) {
    acc = ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__add(acc, buckets[j]);
    res = ag_types__impls__ark_ec__models__short_weierstrass__affine__Affine_ark_bn254__curves__g1__Config__add(res, acc);
  }

  results[gid] = res;
}





